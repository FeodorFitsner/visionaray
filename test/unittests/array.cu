#include "hip/hip_runtime.h"
// This file is distributed under the MIT license.
// See the LICENSE file for details.

#include <algorithm>
#include <cstddef>
#include <cstring> // memcpy

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>

#include <visionaray/array.h>

#include <gtest/gtest.h>

using namespace visionaray;


//-------------------------------------------------------------------------------------------------
// Test reverse iterators
//

template <typename FwdIt, typename T>
__device__ void iota(FwdIt first, FwdIt last, T value)
{
    for (auto it = first; it != last; ++it)
    {
        *it = value++;
    }
}

template <typename Array>
__global__ void kernel_reverse_it(bool* mem, Array /* */)
{
    Array arr1;
    iota(arr1.begin(), arr1.end(), 0);

    // Test non-const iterators for writing
    Array arr2;
    iota(arr2.rbegin(), arr2.rend(), 0);


    size_t i = 0;

    // Test const reverse iterators obtained implicitly through rbegin() and rend()
    auto it1 = arr1.rbegin();
    auto it2 = arr2.begin();
    for (; it1 != arr1.rend() && it2 != arr2.end(); ++it1, ++it2)
    {
        mem[i++] = *it1 == *it2;
    }

    // Test const reverse iterators obtained through crbegin() and crend()
    auto cit1 = arr1.crbegin();
    auto cit2 = arr2.cbegin();
    for (; cit1 != arr1.crend() && cit2 != arr2.cend(); ++cit1, ++cit2)
    {
        mem[i++] = *cit1 == *cit2;
    }
}

TEST(ArrayCU, ReverseIt)
{
    static const size_t N = 50;

    thrust::device_vector<bool> d_result(N * 2);
    thrust::fill(d_result.begin(), d_result.end(), false);

    kernel_reverse_it<<<1, 1>>>(
            thrust::raw_pointer_cast(d_result.data()),
            array<int, N>{}
            );

    thrust::host_vector<bool> h_result(d_result);

    for (auto b : h_result)
    {
        EXPECT_TRUE(b);
    }
}


//-------------------------------------------------------------------------------------------------
// Test array::fill()
//

template <typename T, typename Array>
__global__ void kernel_fill(T value, T* mem, Array /* */)
{
    Array arr;
    arr.fill(value);

    // Copy to global memory so we can compare on the host
    memcpy(mem, arr.data(), sizeof(arr));
}

TEST(ArrayCU, Fill)
{
    static const size_t N = 50;
    thrust::device_vector<int> d_result(N);
    int value = 23;

    kernel_fill<<<1, 1>>>(
            value,
            thrust::raw_pointer_cast(d_result.data()),
            array<int, N>{}
            );

    thrust::host_vector<int> h_result(d_result);

    for (size_t i = 0; i < N; ++i)
    {
        EXPECT_EQ(h_result[i], value);
    }
}


//-------------------------------------------------------------------------------------------------
// Test array::swap()
//

template <typename T, typename Array>
__global__ void kernel_swap(T* mem, Array /* */)
{
    Array arr1;
    Array arr2;

    memcpy(arr1.data(), mem, sizeof(arr1));
    memcpy(arr2.data(), mem + arr1.size(), sizeof(arr2));

    arr1.swap(arr2);

    memcpy(mem, arr1.data(), sizeof(arr1));
    memcpy(mem + arr1.size(), arr2.data(), sizeof(arr2));
}

TEST(ArrayCU, Swap)
{
    static const size_t N = 50;

    thrust::host_vector<int> h_data(N * 2);
    std::fill(h_data.data(), h_data.data() + N, 23);
    std::fill(h_data.data() + N, h_data.data() + h_data.size(), 24);
    thrust::device_vector<int> d_data(h_data);

    kernel_swap<<<1, 1>>>(
            thrust::raw_pointer_cast(d_data.data()),
            array<int, N>{}
            );

    thrust::copy(d_data.begin(), d_data.end(), h_data.begin());

    for (size_t i = 0; i < N; ++i)
    {
        EXPECT_EQ(h_data[i], 24);
    }

    for (size_t i = N; i < N * 2; ++i)
    {
        EXPECT_EQ(h_data[i], 23);
    }

}
